#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

void load_n_launch(hipModule_t& module, int i, int* output, int* output_d){
  hipFunction_t kernel;
  string kernel_name = "list" + to_string(i);
  hipModuleGetFunction(&kernel, module, kernel_name.c_str());

  void * args[] = {&output_d};
  hipModuleLaunchKernel(kernel, 1, 1, 1, 
                 32, 1, 1, 
                 128, 0, args, 0);
  hipDeviceSynchronize();

  hipMemcpy(output, output_d, 32*sizeof(int), hipMemcpyDeviceToHost);

  cout << "nops: \t" << i << ", cycles needed:\t"
       << output[0] << ".\t Cycles per loop:\t" << output[0]/512 << "\n";
}

int main() {
  int * output;
  int * output_d;

  output = (int*)malloc(32*sizeof(int));

  for(int i=0; i<32; ++i) output[i] = 0;

  hipMalloc((void**)&output_d, 32*sizeof(int));
  
  hipModule_t module;
  

  hipModuleLoad(&module, "l1-icache.cubin");
  for(int i=0; i<1024; ++i){
    load_n_launch(module, i, output, output_d);
  }  

  return 0;
  
}
