#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

using namespace std;

void launch_(hipFunction_t kernel, string name, int* output, int* output_d){
  void * args[] = {&output_d};
  hipModuleLaunchKernel(kernel, 1, 1, 1, 
                 32, 1, 1, 
                 128, 0, args, 0);
  hipDeviceSynchronize();

  hipMemcpy(output, output_d, 32*sizeof(int), hipMemcpyDeviceToHost);

  cout << "Cycles to issue 10 " << name << ":     \t"
       << output[0] << ".\n";

  for(int i=0; i<32; ++i) output[i] = 0;
}

int main() {
  int * output;
  int * output_d;

  output = (int*)malloc(32*sizeof(int));

  for(int i=0; i<32; ++i) output[i] = 0;

  hipMalloc((void**)&output_d, 32*sizeof(int));
  
  hipModule_t module;
  hipFunction_t ffma_kernel, fadd_kernel, hadd2_kernel, hfma2_kernel, iadd3_kernel, 
             lea_kernel, imad_kernel, imad_wide_kernel, hmma_kernel;

  hipModuleLoad(&module, "benchmark.cubin");
  hipModuleGetFunction(&ffma_kernel, module, "ffma_cpi");
  hipModuleGetFunction(&fadd_kernel, module, "fadd_cpi");
  hipModuleGetFunction(&hadd2_kernel, module, "hadd2_cpi");
  hipModuleGetFunction(&hfma2_kernel, module, "hfma2_cpi");
  hipModuleGetFunction(&iadd3_kernel, module, "iadd3_cpi");
  hipModuleGetFunction(&lea_kernel, module, "lea_cpi");
  hipModuleGetFunction(&imad_kernel, module, "imad_cpi");
  hipModuleGetFunction(&imad_wide_kernel, module, "imad_wide_cpi");
  hipModuleGetFunction(&hmma_kernel, module, "hmma_cpi");

  launch_(ffma_kernel, "ffma", output, output_d);
  launch_(fadd_kernel, "fadd", output, output_d);  
  launch_(hadd2_kernel, "hadd2", output, output_d);  
  launch_(hfma2_kernel, "hfma2", output, output_d);  
  launch_(iadd3_kernel, "iadd3", output, output_d);
  launch_(lea_kernel, "lea", output, output_d);
  launch_(imad_kernel, "imad", output, output_d);
  launch_(imad_wide_kernel, "imad.wide", output, output_d);
  launch_(hmma_kernel, "hmma.1688.f16", output, output_d);

  hipFunction_t ffma_hadd2_kernel, ffma_imad_kernel, ffma_lea_kernel, ffma_iadd3_kernel,
             hmma_hadd2_kernel, hmma_ffma_kernel, hmma_iadd3_kernel;
  
  hipModuleGetFunction(&ffma_hadd2_kernel, module, "ffma_hadd2");
  hipModuleGetFunction(&ffma_imad_kernel, module, "ffma_imad");
  hipModuleGetFunction(&ffma_lea_kernel, module, "ffma_lea");
  hipModuleGetFunction(&ffma_iadd3_kernel, module, "ffma_iadd3");
  hipModuleGetFunction(&hmma_hadd2_kernel, module, "hmma_hadd2");
  hipModuleGetFunction(&hmma_ffma_kernel, module, "hmma_fmma");
  hipModuleGetFunction(&hmma_iadd3_kernel, module, "hmma_iadd3");

  launch_(ffma_hadd2_kernel, "ffma_hadd2_mix", output, output_d);
  launch_(ffma_imad_kernel, "ffma_imad_mix", output, output_d);
  launch_(ffma_lea_kernel, "ffma_lea_mix", output, output_d);
  launch_(ffma_iadd3_kernel, "ffma_iadd3_mix", output, output_d);
  launch_(hmma_hadd2_kernel, "hmma_hadd2_mix", output, output_d);
  launch_(hmma_ffma_kernel, "hmma_ffma_mix", output, output_d);
  launch_(hmma_iadd3_kernel, "hmma_iadd3_mix", output, output_d);

  return 0;
  
}